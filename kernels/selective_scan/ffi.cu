#include "selective_scan_fwd_kernel.cuh"

extern "C" void selective_scan_fwd_cuda_ffi(
    SSMParamsBase &params,
    uint32_t input_dtype,
    uint32_t weight_dtype,
    hipStream_t stream
    ){
        if (input_dtype == 2 && weight_dtype == 2){
            selective_scan_fwd_cuda<float, float>(params, stream);
        }else{
            exit(1);
        }
}
